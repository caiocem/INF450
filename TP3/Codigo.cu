
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <ctime>

using namespace std;

void initIndex(unsigned long long *index, unsigned long long n)
{
        for(unsigned long long i=0; i<n; i++)
                index[i]=i;
        for(unsigned long long i=0; i<n; i++)
        {
                unsigned long long j = rand()%n;
                swap(index[i],index[j]);
        }
}

void initArray(unsigned long long *a, unsigned long long n)
{
        if(n < (2<<10))
                for(int i = 0; i < n; i++)
                        a[i] = rand()%1001 + 1;
        else
                for(int i = 0; i < n; i++)
                        a[i] = rand()%(n+1) + 1;
}

__global__ void  somaPolinomio(unsigned long long *a, unsigned long long *b, unsigned long long n, unsigned long long *c)
{
        unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;
        if(idx < n)
        {
                unsigned int c1=0,c2=0;
                asm("mov.u32 %0,%%clock;":"=r"(c1));
                #pragma unroll
                for(int i = 0; i < 4; i++)
                {
                        int pos = idx*4+i;
                        c[pos] = 5*(a[pos]*a[pos]*a[pos]) + 7*a[pos]*b[pos] + 8*b[pos]*b[pos] - b[pos] ;
                }
                asm("mov.u32 %0,%%clock;":"=r"(c2));

                if(idx == 0)
                        printf("soma polinomio : %u ms\n",c2-c1);
        }
        return;
}

__global__ void  somaVetor(unsigned long long *a, unsigned long long *b, unsigned long long n, unsigned long long *c)
{
        unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;
        if(idx < n)
        {
                unsigned int c1=0,c2=0;
                asm("mov.u32 %0,%%clock;":"=r"(c1));
                for(int i = 0; i < 4; i++)
                {
                        int pos = idx*4+i;
                        c[pos] = a[pos] + b[pos];
                }
                asm("mov.u32 %0,%%clock;":"=r"(c2));

                if(idx == 0)
                        printf("soma vetor : %u ms\n",c2-c1);
        }
        return;
}

__global__ void  VetorRandom(unsigned long long *a, unsigned long long *b, unsigned long long n, unsigned long long *c, unsigned long long *index)
{
        unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;
        if(idx < n)
        {
                //Acesso Sequencial
                unsigned int c1=0,c2=0;
                asm("mov.u32 %0,%%clock;":"=r"(c1));
                int pos = idx;
                c[pos] = a[pos] + b[pos];
                asm("mov.u32 %0,%%clock;":"=r"(c2));

                //Acesso Aleatorio
                if(idx == 0)
                        printf("Acessa Vetor Sequencial : %u ms\n",c2-c1);

                c1=0;c2=0;
                asm("mov.u32 %0,%%clock;":"=r"(c1));
                c[index[pos]] = a[index[pos]] + b[index[pos]];
                asm("mov.u32 %0,%%clock;":"=r"(c2));

                if(idx == 0)
                        printf("Acessa Vetor Random : %u ms\n",c2-c1);
        }
        return;
}


int main(int argc, char **argv)
{
	srand(time(NULL));
        //TAMANHOS PARA TESTAR : 1M, 2M, 10M, 20M, 32M
        unsigned long long n = 0;
        for(int i = 0; argv[1][i] != '\0'; i++)
                n = n*10 + (argv[1][i]-'0');

        //alocando vetores da CPU
        unsigned long long * h_a = new unsigned long long[n];
        unsigned long long * h_b = new unsigned long long[n];
        unsigned long long * h_c = new unsigned long long[n];
        unsigned long long * h_index = new unsigned long long[n];
        initArray(h_a,n);
        initArray(h_b,n);
        initIndex(h_index,n);

        //alocando vetores
        unsigned long long * d_a, *d_b, *d_c;
        hipMalloc(&d_a,sizeof(unsigned long long)*n);
        hipMalloc(&d_b,sizeof(unsigned long long)*n);
        hipMalloc(&d_c,sizeof(unsigned long long)*n);
        unsigned long long * d_index;
        hipMalloc(&d_index,sizeof(unsigned long long)*n);

	//copiando valores da CPU para a GPU
        hipMemcpy(d_a, h_a, sizeof(unsigned long long)*n,hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, sizeof(unsigned long long)*n,hipMemcpyHostToDevice);
        hipMemcpy(d_index, h_index, sizeof(unsigned long long)*n,hipMemcpyHostToDevice);

        dim3 block,grid;
        //tamanho do bloco é arbitrário
        block.x = 1024;

        grid.x = ((n/4 + block.x -1)/block.x);

        //chama o somaVetor
        somaVetor<<<grid,block>>>(d_a,d_b,n,d_c);
        hipDeviceSynchronize();

        //traz o resultado da GPU para a CPU
        hipMemcpy(h_c, d_c, sizeof(unsigned long long)*n,hipMemcpyDeviceToHost);

        //debug somaVetor
        /*for(int i = 0; i < n; i++)
                cout << h_a[i] << " ";
        cout << "\n";
        for(int i = 0; i < n; i++)
                cout << h_b[i] << " ";
        cout << "\n";
        for(int i = 0; i < n; i++)
                cout << h_c[i] << " ";
        cout << "\n";*/
	
	//chama o somaPolinomio
        somaPolinomio<<<grid,block>>>(d_a,d_b,n,d_c);
        hipDeviceSynchronize();

        //chama VetorRandom
        VetorRandom<<<grid,block>>>(d_a,d_b,n,d_c,d_index);
        hipDeviceSynchronize();


        hipMemcpy(h_c, d_c, sizeof(unsigned long long)*n,hipMemcpyDeviceToHost);

        //debug somaVetor
        /*for(int i = 0; i < n; i++)
                cout << h_a[i] << " ";
        cout << "\n";
        for(int i = 0; i < n; i++)
                cout << h_b[i] << " ";
        cout << "\n";
        for(int i = 0; i < n; i++)
                cout << h_c[i] << " ";
        cout << "\n";*/

        //desalocando memória
        free(h_a);
        free(h_b);
	free(h_c);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        //reseta a GPU
        hipDeviceReset();
        return 0;
}

